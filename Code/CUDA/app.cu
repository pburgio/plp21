#include <stdio.h>
#include <hip/hip_runtime.h>

int arr[15];

__global__ void funzione()
{
  //printf("\t\t\t\t\t[DEVICE] Hello World!\n");
  
  int thrId = threadIdx.x;
  int blkId = blockIdx.x;
  
  int thrNum = blockDim.x;
  int blkNum = gridDim.x;

  int i =  threadIdx.x + blockIdx.x * blockDim.x;
  // every thread has a unique i
  if(i >= 6)
    return;

  arr[i] = i*2;// qualcosa
  
  printf("\t\t\t\t\t[DEVICE] Hello World! I am thread #%d out of %d, and I belong to block #%d out of %d\n", thrId, thrNum, blkId, blkNum);
  
  return;
}

int main()
{
  printf("[HOST] Hello World!\n");
  
  funzione<<<3,5>>>();

  printf("[HOST] Some more work on host\n");

  hipDeviceSynchronize();
  
  printf("[HOST] Device ended its work!\n");
  
  return 0;
}
